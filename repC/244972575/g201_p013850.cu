#include "hip/hip_runtime.h"
// User: g201@79.109.79.14 
// ExecutionRequest[P:'erCho.cu',P:1,T:1,args:'',q:'cudalb'] 
// May 16 2019 18:03:51
#include "cputils.h" // Added by tablon
/*30 30 100 2 9 18 2 29 26 3 2 3 6 4 800 25 20 2 900
 * Simplified simulation of fire extinguishing
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2018/2019
 *
 * v1.4
 *
 * (c) 2019 Arturo Gonzalez Escribano
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <cputils.h>

#define RADIUS_TYPE_1		3
#define RADIUS_TYPE_2_3		9
#define THRESHOLD	0.1f

/* Structure to store data of an extinguishing team */
typedef struct {
	int x,y;
	int type;
	int target;
} Team;

/* Structure to store data of a fire focal point */
typedef struct {
	int x,y;
	int start;
	int heat;
	int active; // States: 0 Not yet activated; 1 Active; 2 Deactivated by a team
} FocalPoint;

/* Macro function to simplify accessing with two coordinates to a flattened array */
#define accessMat( arr, exp1, exp2 )	arr[ (exp1) * columns + (exp2) ]

__global__ void init(float *surface,int rows,int columns){

	int idX=threadIdx.y+blockDim.y*blockIdx.y;
	int idY=threadIdx.x+blockDim.x*blockIdx.x;

	if (idX >= rows || idY>= columns) return;

	surface[idX*columns+idY]=0;


}

__global__ void initInt(int *surface, int rows, int columns){

	int idX=threadIdx.y+blockDim.y*blockIdx.y;
	int idY=threadIdx.x+blockDim.x*blockIdx.x;

	if (idX >= rows || idY>= columns) return;

	surface[idX*columns+idY]=0;
}

__global__ void get_first_activation(FocalPoint *focal, int num_focal,int *salida){

	__shared__ int first_activation;

	int id=threadIdx.x+blockDim.x*blockIdx.x;
	if(id>=num_focal) return;
	first_activation=0;


	atomicMin(&first_activation,focal[id].start);

	__syncthreads();
	if(id==0)
		salida[0]=first_activation;
}

__global__ void activate_focal(FocalPoint *focal,int num_focal,int *salida,int iter){

	__shared__ int num_deactivated;
	int id=threadIdx.x+blockDim.x*blockIdx.x;


	if(id>=num_focal) return;
	num_deactivated=0;


//printf("iter hilo %d num_ %d\n",iter,num_deactivated );
	if ( focal[id].active == 2 ) {
		atomicAdd(&num_deactivated,1);


	}
		if ( focal[id].start == iter ) {
			focal[id].active = 1;

		}
			__syncthreads();
		if(id==0)
		salida[0]=num_deactivated;
		// Count focal points already deactivated by a team



}
__global__ void update_heat(float *surface,FocalPoint *focal, int columns , int num_focal){

		int id=threadIdx.x+blockDim.x*blockIdx.x;
		if(id>=num_focal || focal[id].active!=1) return;

	surface[focal[id].x*columns+focal[id].y]=focal[id].heat;
}

__global__ void copy_surface(float *surface, float *surfaceCopy,int rows,int columns){

	int idX=threadIdx.y+blockDim.y*blockIdx.y;
	int idY=threadIdx.x+blockDim.x*blockIdx.x;

	if (idX >= rows-1 || idX==0 || idY>= columns-1 || idY==0) return;

	surfaceCopy[idX*columns+idY]=surface[idX*columns+idY];

}

__global__ void update_surface(float *surface, float *surfaceCopy,int rows, int columns){
	int idX=threadIdx.y+blockDim.y*blockIdx.y;
	int idY=threadIdx.x+blockDim.x*blockIdx.x;
  //printf("hola\n" );
	if (idX >= rows-1 || idX==0 || idY>= columns-1 || idY==0) return;

	surface[idX*columns+idY]=(
		surfaceCopy[(idX-1)*columns+idY]+
		surfaceCopy[(idX+1)*columns+idY]+
		surfaceCopy[idX*columns+idY-1]+
		surfaceCopy[idX*columns+idY+1])/4;
	//printf("%f",surface[idX*columns+idY]);
	/*int i, j;
	for( i=1; i<rows-1; i++ )
		for( j=1; j<columns-1; j++ )
			accessMat( surface, i, j ) = (
				accessMat( surfaceCopy, i-1, j ) +
				accessMat( surfaceCopy, i+1, j ) +
				accessMat( surfaceCopy, i, j-1 ) +
				accessMat( surfaceCopy, i, j+1 ) ) / 4;*/
}

__global__ void compute_residual(float *surface, float *surfaceCopy,int rows,int columns,float *residuals){

	int idX=threadIdx.y+blockDim.y*blockIdx.y;
	int idY=threadIdx.x+blockDim.x*blockIdx.x;
	//printf("hola\n" );
	//if (idX >= rows-1 || idX==0 || idY>= columns-1 || idY==0) return;
	if(idX>=rows || idY>=columns) return;
	residuals[idX*columns+idY]=surface[idX*columns+idY]-surfaceCopy[idX*columns+idY];
}



__global__ void move_teams(Team *teams,FocalPoint *focal, int num_teams,int num_focal){

		int id=threadIdx.x+blockDim.x*blockIdx.x;

		if(id>=num_teams) return;
		unsigned int j;
		int distance = INT_MAX;
		int target = -1;
		int teamX = teams[id].x;
		int teamY = teams[id].y;
		#pragma unroll
		for( j=0; j<num_focal; j++ ) {
			if ( focal[j].active != 1 ) continue; // Skip non-active focal points

			int local_distance =  (focal[j].x - teamX)*(focal[j].x - teamX) + (focal[j].y - teamY)*(focal[j].y - teamY) ;
			if ( local_distance < distance ) {
				distance = local_distance;
				target = j;
			}
		}
		/* 4.3.2. Annotate target for the next stage */
		teams[id].target = target;

		/* 4.3.3. No active focal point to choose, no movement */
		if ( target == -1 ) return;
		//__syncthreads();
		/* 4.3.4. Move in the focal point direction */

		int focalX = focal[target].x;
		int focalY = focal[target].y;
		if ( teams[id].type == 1 ) {
			// Type 1: Can move in diagonal
			if ( focalX < teams[id].x ) teams[id].x--;
			if ( focalX > teams[id].x ) teams[id].x++;
			if ( focalY < teams[id].y ) teams[id].y--;
			if ( focalY > teams[id].y) teams[id].y++;
		}
		else if ( teams[id].type == 2 ) {
			// Type 2: First in horizontal direction, then in vertical direction
			if ( focalY < teamY ) teams[id].y--;
			else if ( focalY > teamY ) teams[id].y++;
			else if ( focalX < teamX ) teams[id].x--;
			else if ( focalX > teamX ) teams[id].x++;
		}
		else {
			// Type 3: First in vertical direction, then in horizontal direction
			if ( focalX < teamX ) teams[id].x--;
			else if ( focalX > teamX ) teams[id].x++;
			else if ( focalY < teamY ) teams[id].y--;
			else if ( focalY > teamY ) teams[id].y++;
		}

		//printf("x %d y %d id %d\n", teams[id].x,teams[id].y,id);
		if ( target != -1 && focalX == teams[id].x && focalY == teams[id].y
			&& focal[target].active == 1 ){
			focal[target].active = 2;
			//printf("id %d\n",id);
		}
}

__global__ void compute_heat_reduction(Team *teams,int *gpuAux,int num_teams,int rows,int columns){

	int id=threadIdx.x+blockDim.x*blockIdx.x;
	if(id>=num_teams) return;
	//int radius;

	// Influence area of fixed radius depending on type
	//if ( teams[id].type == 1 ) radius = 3;
	//else radius = 9;
	int teamX=teams[id].x;
	int teamY=teams[id].y;
	//#pragma unroll
	//for( i=teams[id].x-radius; i<=teams[id].x+radius; i++ ) {
		//#pragma unroll
		//for( j=teams[id].y-radius; j<=teams[id].y+radius; j++ ) {
		if (teams[id].type!=1){

			if ( (teamX-9)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-9)*columns+teamY],1);

			if ( (teamX-8)>0 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY-4],1);
			if ( (teamX-8)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY-3],1);
			if ( (teamX-8)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY-2],1);
			if ( (teamX-8)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY-1],1);
			if ( (teamX-8)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY],1);

			if ( (teamX-8)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY+1],1);
			if ( (teamX-8)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY+2],1);
			if ( (teamX-8)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY+3],1);
			if ( (teamX-8)>0 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX-8)*columns+teamY+4],1);

			if ( (teamX-7)>0 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY-5],1);
			if ( (teamX-7)>0 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY-4],1);
			if ( (teamX-7)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY-3],1);
			if ( (teamX-7)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY-2],1);
			if ( (teamX-7)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY-1],1);
			if ( (teamX-7)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY],1);

			if ( (teamX-7)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY+1],1);
			if ( (teamX-7)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY+2],1);
			if ( (teamX-7)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY+3],1);
			if ( (teamX-7)>0 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY+4],1);
			if ( (teamX-7)>0 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX-7)*columns+teamY+5],1);

			if ( (teamX-6)>0 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY-6],1);
			if ( (teamX-6)>0 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY-5],1);
			if ( (teamX-6)>0 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY-4],1);
			if ( (teamX-6)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY-3],1);
			if ( (teamX-6)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY-2],1);
			if ( (teamX-6)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY-1],1);
			if ( (teamX-6)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY],1);

			if ( (teamX-6)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY+1],1);
			if ( (teamX-6)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY+2],1);
			if ( (teamX-6)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY+3],1);
			if ( (teamX-6)>0 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY+4],1);
			if ( (teamX-6)>0 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY+5],1);
			if ( (teamX-6)>0 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX-6)*columns+teamY+6],1);

			if ( (teamX-5)>0 && (teamY-7)>0 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY-7],1);
			if ( (teamX-5)>0 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY-6],1);
			if ( (teamX-5)>0 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY-5],1);
			if ( (teamX-5)>0 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY-4],1);
			if ( (teamX-5)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY-3],1);
			if ( (teamX-5)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY-2],1);
			if ( (teamX-5)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY-1],1);
			if ( (teamX-5)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY],1);

			if ( (teamX-5)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY+1],1);
			if ( (teamX-5)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY+2],1);
			if ( (teamX-5)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY+3],1);
			if ( (teamX-5)>0 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY+4],1);
			if ( (teamX-5)>0 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY+5],1);
			if ( (teamX-5)>0 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY+6],1);
			if ( (teamX-5)>0 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[(teamX-5)*columns+teamY+7],1);

			if ( (teamX-4)>0 && (teamY-8)>0 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY-8],1);
			if ( (teamX-4)>0 && (teamY-7)>0 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY-7],1);
			if ( (teamX-4)>0 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY-6],1);
			if ( (teamX-4)>0 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY-5],1);
			if ( (teamX-4)>0 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY-4],1);
			if ( (teamX-4)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY-3],1);
			if ( (teamX-4)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY-2],1);
			if ( (teamX-4)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY-1],1);
			if ( (teamX-4)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY],1);

			if ( (teamX-4)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY+1],1);
			if ( (teamX-4)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY+2],1);
			if ( (teamX-4)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY+3],1);
			if ( (teamX-4)>0 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY+4],1);
			if ( (teamX-4)>0 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY+5],1);
			if ( (teamX-4)>0 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY+6],1);
			if ( (teamX-4)>0 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY+7],1);
			if ( (teamX-4)>0 && (teamY+8)<columns-1 )
			atomicAdd(&gpuAux[(teamX-4)*columns+teamY+8],1);

			if ( (teamX-3)>0 && (teamY-8)>0 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY-8],1);
			if ( (teamX-3)>0 && (teamY-7)>0 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY-7],1);
			if ( (teamX-3)>0 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY-6],1);
			if ( (teamX-3)>0 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY-5],1);
			if ( (teamX-3)>0 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY-4],1);
			if ( (teamX-3)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY-3],1);
			if ( (teamX-3)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY-2],1);
			if ( (teamX-3)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY-1],1);
			if ( (teamX-3)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY],1);

			if ( (teamX-3)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY+1],1);
			if ( (teamX-3)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY+2],1);
			if ( (teamX-3)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY+3],1);
			if ( (teamX-3)>0 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY+4],1);
			if ( (teamX-3)>0 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY+5],1);
			if ( (teamX-3)>0 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY+6],1);
			if ( (teamX-3)>0 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY+7],1);
			if ( (teamX-3)>0 && (teamY+8)<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY+8],1);

			if ( (teamX-2)>0 && (teamY-8)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-8],1);
			if ( (teamX-2)>0 && (teamY-7)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-7],1);
			if ( (teamX-2)>0 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-6],1);
			if ( (teamX-2)>0 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-5],1);
			if ( (teamX-2)>0 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-4],1);
			if ( (teamX-2)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-3],1);
			if ( (teamX-2)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-2],1);
			if ( (teamX-2)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-1],1);
			if ( (teamX-2)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY],1);

			if ( (teamX-2)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+1],1);
			if ( (teamX-2)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+2],1);
			if ( (teamX-2)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+3],1);
			if ( (teamX-2)>0 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+4],1);
			if ( (teamX-2)>0 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+5],1);
			if ( (teamX-2)>0 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+6],1);
			if ( (teamX-2)>0 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+7],1);
			if ( (teamX-2)>0 && (teamY+8)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+8],1);
	    if ( (teamX-1)>0 && (teamY-8)>0 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY-8],1);
	    if ( (teamX-1)>0 && (teamY-7)>0 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY-7],1);
	    if ( (teamX-1)>0 && (teamY-6)>0 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY-6],1);
	    if ( (teamX-1)>0 && (teamY-5)>0 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY-5],1);
	    if ( (teamX-1)>0 && (teamY-4)>0 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY-4],1);
	    if ( (teamX-1)>0 && (teamY-3)>0 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY-3],1);
	    if ( (teamX-1)>0 && (teamY-2)>0 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY-2],1);
	    if ( (teamX-1)>0 && (teamY-1)>0 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY-1],1);
	    if ( (teamX-1)>0 && (teamY)>0 && teamY<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY],1);

	    if ( (teamX-1)>0 && (teamY+1)<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY+1],1);
	    if ( (teamX-1)>0 && (teamY+2)<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY+2],1);
	    if ( (teamX-1)>0 && (teamY+3)<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY+3],1);
	    if ( (teamX-1)>0 && (teamY+4)<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY+4],1);
	    if ( (teamX-1)>0 && (teamY+5)<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY+5],1);
	    if ( (teamX-1)>0 && (teamY+6)<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY+6],1);
	    if ( (teamX-1)>0 && (teamY+7)<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY+7],1);
	    if ( (teamX-1)>0 && (teamY+8)<columns-1 )
	    atomicAdd(&gpuAux[(teamX-1)*columns+teamY+8],1);


			if ( (teamX)>0 && (teamY-9)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-9],1);
			if ( (teamX)>0 && (teamY-8)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-8],1);
			if ( (teamX)>0 && (teamY-7)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-7],1);
			if ( (teamX)>0 && (teamY-6)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-6],1);
			if ( (teamX)>0 && (teamY-5)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-5],1);
			if ( (teamX)>0 && (teamY-4)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-4],1);
			if ( (teamX)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-3],1);
			if ( (teamX)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-2],1);
			if ( (teamX)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-1],1);
			if ( (teamX)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY],1);

			if ( (teamX)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+1],1);
			if ( (teamX)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+2],1);
			if ( (teamX)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+3],1);
			if ( (teamX)>0 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+4],1);
			if ( (teamX)>0 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+5],1);
			if ( (teamX)>0 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+6],1);
			if ( (teamX)>0 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+7],1);
			if ( (teamX)>0 && (teamY+8)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+8],1);
			if ( (teamX)>0 && (teamY+9)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+9],1);


	    if ( (teamX+1)<rows-1 && (teamY-8)>0 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY-8],1);
	    if ( (teamX+1)<rows-1 && (teamY-7)>0 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY-7],1);
	    if ( (teamX+1)<rows-1 && (teamY-6)>0 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY-6],1);
	    if ( (teamX+1)<rows-1 && (teamY-5)>0 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY-5],1);
	    if ( (teamX+1)<rows-1 && (teamY-4)>0 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY-4],1);
	    if ( (teamX+1)<rows-1 && (teamY-3)>0 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY-3],1);
	    if ( (teamX+1)<rows-1 && (teamY-2)>0 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY-2],1);
	    if ( (teamX+1)<rows-1 && (teamY-1)>0 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY-1],1);
	    if ( (teamX+1)<rows-1 && (teamY)>0 && teamY<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY],1);

	    if ( (teamX+1)<rows-1 && (teamY+1)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY+1],1);
	    if ( (teamX+1)<rows-1 && (teamY+2)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY+2],1);
	    if ( (teamX+1)<rows-1 && (teamY+3)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY+3],1);
	    if ( (teamX+1)<rows-1 && (teamY+4)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY+4],1);
	    if ( (teamX+1)<rows-1 && (teamY+5)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY+5],1);
	    if ( (teamX+1)<rows-1 && (teamY+6)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY+6],1);
	    if ( (teamX+1)<rows-1 && (teamY+7)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY+7],1);
	    if ( (teamX+1)<rows-1 && (teamY+8)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+1)*columns+teamY+8],1);



			if ( (teamX+2)<rows-1 && (teamY-8)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-8],1);
			if ( (teamX+2)<rows-1 && (teamY-7)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-7],1);
			if ( (teamX+2)<rows-1 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-6],1);
			if ( (teamX+2)<rows-1 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-5],1);
			if ( (teamX+2)<rows-1 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-4],1);
			if ( (teamX+2)<rows-1 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-3],1);
			if ( (teamX+2)<rows-1 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-2],1);
			if ( (teamX+2)<rows-1 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-1],1);
			if ( (teamX+2)<rows-1 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY],1);

			if ( (teamX+2)<rows-1 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+1],1);
			if ( (teamX+2)<rows-1 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+2],1);
			if ( (teamX+2)<rows-1 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+3],1);
			if ( (teamX+2)<rows-1 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+4],1);
			if ( (teamX+2)<rows-1 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+5],1);
			if ( (teamX+2)<rows-1 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+6],1);
			if ( (teamX+2)<rows-1 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+7],1);
			if ( (teamX+2)<rows-1 && (teamY+8)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+8],1);



			if ( (teamX+3)<rows-1 && (teamY-8)>0 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY-8],1);
			if ( (teamX+3)<rows-1 && (teamY-7)>0 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY-7],1);
			if ( (teamX+3)<rows-1 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY-6],1);
			if ( (teamX+3)<rows-1 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY-5],1);
			if ( (teamX+3)<rows-1 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY-4],1);
			if ( (teamX+3)<rows-1 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY-3],1);
			if ( (teamX+3)<rows-1 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY-2],1);
			if ( (teamX+3)<rows-1 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY-1],1);
			if ( (teamX+3)<rows-1 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY],1);

			if ( (teamX+3)<rows-1 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY+1],1);
			if ( (teamX+3)<rows-1 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY+2],1);
			if ( (teamX+3)<rows-1 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY+3],1);
			if ( (teamX+3)<rows-1 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY+4],1);
			if ( (teamX+3)<rows-1 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY+5],1);
			if ( (teamX+3)<rows-1 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY+6],1);
			if ( (teamX+3)<rows-1 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY+7],1);
			if ( (teamX+3)<rows-1 && (teamY+8)<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY+8],1);



			if ( (teamX+4)<rows-1 && (teamY-8)>0 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY-8],1);
			if ( (teamX+4)<rows-1 && (teamY-7)>0 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY-7],1);
			if ( (teamX+4)<rows-1 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY-6],1);
			if ( (teamX+4)<rows-1 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY-5],1);
			if ( (teamX+4)<rows-1 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY-4],1);
			if ( (teamX+4)<rows-1 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY-3],1);
			if ( (teamX+4)<rows-1 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY-2],1);
			if ( (teamX+4)<rows-1 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY-1],1);
			if ( (teamX+4)<rows-1 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY],1);

			if ( (teamX+4)<rows-1 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY+1],1);
			if ( (teamX+4)<rows-1 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY+2],1);
			if ( (teamX+4)<rows-1 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY+3],1);
			if ( (teamX+4)<rows-1 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY+4],1);
			if ( (teamX+4)<rows-1 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY+5],1);
			if ( (teamX+4)<rows-1 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY+6],1);
			if ( (teamX+4)<rows-1 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY+7],1);
			if ( (teamX+4)<rows-1 && (teamY+8)<columns-1 )
			atomicAdd(&gpuAux[(teamX+4)*columns+teamY+8],1);


			if ( (teamX+5)<rows-1 && (teamY-7)>0 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY-7],1);
			if ( (teamX+5)<rows-1 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY-6],1);
			if ( (teamX+5)<rows-1 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY-5],1);
			if ( (teamX+5)<rows-1 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY-4],1);
			if ( (teamX+5)<rows-1 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY-3],1);
			if ( (teamX+5)<rows-1 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY-2],1);
			if ( (teamX+5)<rows-1 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY-1],1);
			if ( (teamX+5)<rows-1 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY],1);

			if ( (teamX+5)<rows-1 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY+1],1);
			if ( (teamX+5)<rows-1 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY+2],1);
			if ( (teamX+5)<rows-1 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY+3],1);
			if ( (teamX+5)<rows-1 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY+4],1);
			if ( (teamX+5)<rows-1 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY+5],1);
			if ( (teamX+5)<rows-1 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY+6],1);
			if ( (teamX+5)<rows-1 && (teamY+7)<columns-1 )
			atomicAdd(&gpuAux[(teamX+5)*columns+teamY+7],1);



			if ( (teamX+6)<rows-1 && (teamY-6)>0 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY-6],1);
			if ( (teamX+6)<rows-1 && (teamY-5)>0 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY-5],1);
			if ( (teamX+6)<rows-1 && (teamY-4)>0 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY-4],1);
			if ( (teamX+6)<rows-1 && (teamY-3)>0 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY-3],1);
			if ( (teamX+6)<rows-1 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY-2],1);
			if ( (teamX+6)<rows-1 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY-1],1);
			if ( (teamX+6)<rows-1 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY],1);

			if ( (teamX+6)<rows-1 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY+1],1);
			if ( (teamX+6)<rows-1 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY+2],1);
			if ( (teamX+6)<rows-1 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY+3],1);
			if ( (teamX+6)<rows-1 && (teamY+4)<columns-1 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY+4],1);
			if ( (teamX+6)<rows-1 && (teamY+5)<columns-1 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY+5],1);
			if ( (teamX+6)<rows-1 && (teamY+6)<columns-1 )
			atomicAdd(&gpuAux[(teamX+6)*columns+teamY+6],1);


	    if ( (teamX+7)<rows-1 && (teamY-5)>0 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY-5],1);
	    if ( (teamX+7)<rows-1 && (teamY-4)>0 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY-4],1);
	    if ( (teamX+7)<rows-1 && (teamY-3)>0 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY-3],1);
	    if ( (teamX+7)<rows-1 && (teamY-2)>0 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY-2],1);
	    if ( (teamX+7)<rows-1 && (teamY-1)>0 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY-1],1);
	    if ( (teamX+7)<rows-1 && (teamY)>0 && teamY<columns-1 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY],1);

	    if ( (teamX+7)<rows-1 && (teamY+1)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY+1],1);
	    if ( (teamX+7)<rows-1 && (teamY+2)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY+2],1);
	    if ( (teamX+7)<rows-1 && (teamY+3)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY+3],1);
	    if ( (teamX+7)<rows-1 && (teamY+4)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY+4],1);
	    if ( (teamX+7)<rows-1 && (teamY+5)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+7)*columns+teamY+5],1);


	    if ( (teamX+8)<rows-1 && (teamY-4)>0 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY-4],1);
	    if ( (teamX+8)<rows-1 && (teamY-3)>0 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY-3],1);
	    if ( (teamX+8)<rows-1 && (teamY-2)>0 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY-2],1);
	    if ( (teamX+8)<rows-1 && (teamY-1)>0 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY-1],1);
	    if ( (teamX+8)<rows-1 && (teamY)>0 && teamY<columns-1 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY],1);

	    if ( (teamX+8)<rows-1 && (teamY+1)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY+1],1);
	    if ( (teamX+8)<rows-1 && (teamY+2)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY+2],1);
	    if ( (teamX+8)<rows-1 && (teamY+3)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY+3],1);
	    if ( (teamX+8)<rows-1 && (teamY+4)<columns-1 )
	    atomicAdd(&gpuAux[(teamX+8)*columns+teamY+4],1);


	    if ( (teamX+9)<rows-1 && (teamY)>0 && teamY<columns-1 )
	    atomicAdd(&gpuAux[(teamX+9)*columns+teamY],1);





		}

		else{


			if ( (teamX-3)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-3)*columns+teamY],1);




			if ( (teamX-2)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-2],1);
			if ( (teamX-2)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY-1],1);
			if ( (teamX-2)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY],1);

			if ( (teamX-2)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+1],1);
			if ( (teamX-2)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-2)*columns+teamY+2],1);




			if ( (teamX-1)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX-1)*columns+teamY-2],1);
			if ( (teamX-1)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX-1)*columns+teamY-1],1);
			if ( (teamX-1)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX-1)*columns+teamY],1);

			if ( (teamX-1)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX-1)*columns+teamY+1],1);
			if ( (teamX-1)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX-1)*columns+teamY+2],1);




			if ( (teamX)>0 && (teamY-3)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-3],1);
			if ( (teamX)>0 && (teamY-2)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-2],1);
			if ( (teamX)>0 && (teamY-1)>0 )
			atomicAdd(&gpuAux[teamX*columns+teamY-1],1);
			if ( (teamX)>0 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY],1);

			if ( (teamX)>0 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+1],1);
			if ( (teamX)>0 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+2],1);
			if ( (teamX)>0 && (teamY+3)<columns-1 )
			atomicAdd(&gpuAux[teamX*columns+teamY+3],1);





			if ( (teamX+1)<rows-1 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX+1)*columns+teamY-2],1);
			if ( (teamX+1)<rows-1 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX+1)*columns+teamY-1],1);
			if ( (teamX+1)<rows-1 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX+1)*columns+teamY],1);

			if ( (teamX+1)<rows-1 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX+1)*columns+teamY+1],1);
			if ( (teamX+1)<rows-1 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX+1)*columns+teamY+2],1);






			if ( (teamX+2)<rows-1 && (teamY-2)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-2],1);
			if ( (teamX+2)<rows-1 && (teamY-1)>0 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY-1],1);
			if ( (teamX+2)<rows-1 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY],1);

			if ( (teamX+2)<rows-1 && (teamY+1)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+1],1);
			if ( (teamX+2)<rows-1 && (teamY+2)<columns-1 )
			atomicAdd(&gpuAux[(teamX+2)*columns+teamY+2],1);





			if ( (teamX+3)<rows-1 && (teamY)>0 && teamY<columns-1 )
			atomicAdd(&gpuAux[(teamX+3)*columns+teamY],1);



		}

}

__global__ void reduce_heat3(float *surface, int *aux,int rows,int columns){

	int idX=threadIdx.y+blockDim.y*blockIdx.y;
	int idY=threadIdx.x+blockDim.x*blockIdx.x;
  //printf("hola\n" );
	if (idX >= rows-1 || idX==0 || idY>= columns-1 || idY==0) return;
	#pragma unroll
	for(unsigned int i=aux[idX*columns+idY];i>0;i--)
		surface[idX*columns+idY]*=0.75;

	aux[idX*columns+idY]=0;
}



__global__ void reduce_kernel(const float* g_idata, float* g_odata, int size)
{
	// Memoria shared
	extern __shared__ float tmp[];

	// Desactivar hilos que excedan los límites del array de entrada
	int gid = threadIdx.x+blockDim.x*blockIdx.x;
   if ( gid >= size ) return;

	// Cargar dato en memoria shared
	int tid = threadIdx.x;
	tmp[ tid ] = g_idata[ gid ];
//printf("entrada  %f glob red %f\n",g_idata[gid],tmp[tid]);
	// Asegurarse que todos los warps del bloque han cargado los datos
	__syncthreads();

	// Generalización: El único bloque del último nivel puede tener menos datos para reducir
	int mysize = blockDim.x;
	if ( gridDim.x==1 )
		mysize = size;

	// Hacemos la reducción en memoria shared
	#pragma unroll
	for(unsigned int s = mysize/2; s >0; s /= 2) {
		// Comprobamos si el hilo actual es activo para esta iteración

		if (tid<s) {
			// Hacemos la reducción sumando los dos elementos que le tocan a este hilo
			if(tmp[tid+s]>tmp[tid])
			tmp[tid]  =tmp[tid+s];
		}
		__syncthreads();
	}

	// El hilo 0 de cada bloque escribe el resultado final de la reducción
	// en la memoria global del dispositivo pasada por parámetro (g_odata[])
	if (tid == 0){
		g_odata[blockIdx.x] = tmp[tid];

	}
}
/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s <config_file> | <command_line_args>\n", program_name );
	fprintf(stderr,"\t<config_file> ::= -f <file_name>\n");
	fprintf(stderr,"\t<command_line_args> ::= <rows> <columns> <maxIter> <numTeams> [ <teamX> <teamY> <teamType> ... ] <numFocalPoints> [ <focalX> <focalY> <focalStart> <focalTemperature> ... ]\n");
	fprintf(stderr,"\n");
}

#ifdef DEBUG
/*
 * Function: Print the current state of the simulation
 */
void print_status( int iteration, int rows, int columns, float *surface, int num_teams, Team *teams, int num_focal, FocalPoint *focal, float global_residual ) {
	/*
	 * You don't need to optimize this function, it is only for pretty printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;

	printf("Iteration: %d\n", iteration );
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( surface, i, j ) >= 1000 ) symbol = '*';
			else if ( accessMat( surface, i, j ) >= 100 ) symbol = '0' + (int)(accessMat( surface, i, j )/100);
			else if ( accessMat( surface, i, j ) >= 50 ) symbol = '+';
			else if ( accessMat( surface, i, j ) >= 25 ) symbol = '.';
			else symbol = '0';

			int t;
			int flag_team = 0;
			for( t=0; t<num_teams; t++ )
				if ( teams[t].x == i && teams[t].y == j ) { flag_team = 1; break; }
			if ( flag_team ) printf("[%c]", symbol );
			else {
				int f;
				int flag_focal = 0;
				for( f=0; f<num_focal; f++ )
					if ( focal[f].x == i && focal[f].y == j && focal[f].active == 1 ) { flag_focal = 1; break; }
				if ( flag_focal ) printf("(%c)", symbol );
				else printf(" %c ", symbol );
			}
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	printf("Global residual: %f\n\n", global_residual);
}
#endif

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	int i,j,t;

	// Simulation data
	int rows, columns, max_iter;
	float *surface, *surfaceCopy;
	int num_teams, num_focal;
	Team *teams;
	FocalPoint *focal;


	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc<2) {
		fprintf(stderr,"-- Error in arguments: No arguments\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	int read_from_file = ! strcmp( argv[1], "-f" );
	/* 1.2. Read configuration from file */
	if ( read_from_file ) {
		/* 1.2.1. Open file */
		if (argc<3) {
			fprintf(stderr,"-- Error in arguments: file-name argument missing\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		FILE *args = cp_abrir_fichero( argv[2] );
		if ( args == NULL ) {
			fprintf(stderr,"-- Error in file: not found: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}

		/* 1.2.2. Read surface and maximum number of iterations */
		int ok;
		ok = fscanf(args, "%d %d %d", &rows, &columns, &max_iter);
		if ( ok != 3 ) {
			fprintf(stderr,"-- Error in file: reading rows, columns, max_iter from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		if ( surface == NULL || surfaceCopy == NULL ) {
			fprintf(stderr,"-- Error allocating: surface structures\n");
			exit( EXIT_FAILURE );
		}

		/* 1.2.3. Teams information */
		ok = fscanf(args, "%d", &num_teams );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error file, reading num_teams from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			ok = fscanf(args, "%d %d %d", &teams[i].x, &teams[i].y, &teams[i].type);
			if ( ok != 3 ) {
				fprintf(stderr,"-- Error in file: reading team %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
		}

		/* 1.2.4. Focal points information */
		ok = fscanf(args, "%d", &num_focal );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error in file: reading num_focal from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( focal == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			ok = fscanf(args, "%d %d %d %d", &focal[i].x, &focal[i].y, &focal[i].start, &focal[i].heat);
			if ( ok != 4 ) {
				fprintf(stderr,"-- Error in file: reading focal point %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
			focal[i].active = 0;
		}
	}
	/* 1.3. Read configuration from arguments */
	else {
		/* 1.3.1. Check minimum number of arguments */
		if (argc<6) {
			fprintf(stderr, "-- Error in arguments: not enough arguments when reading configuration from the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}

		/* 1.3.2. Surface and maximum number of iterations */
		rows = atoi( argv[1] );
		columns = atoi( argv[2] );
		max_iter = atoi( argv[3] );

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		/* 1.3.3. Teams information */
		num_teams = atoi( argv[4] );
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		if ( argc < num_teams*3 + 5 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			teams[i].x = atoi( argv[5+i*3] );
			teams[i].y = atoi( argv[6+i*3] );
			teams[i].type = atoi( argv[7+i*3] );
		}

		/* 1.3.4. Focal points information */
		int focal_args = 5 + i*3;
		if ( argc < focal_args+1 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for the number of focal points\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		num_focal = atoi( argv[focal_args] );
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		if ( argc < focal_args + 1 + num_focal*4 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			focal[i].x = atoi( argv[focal_args+i*4+1] );
			focal[i].y = atoi( argv[focal_args+i*4+2] );
			focal[i].start = atoi( argv[focal_args+i*4+3] );
			focal[i].heat = atoi( argv[focal_args+i*4+4] );
			focal[i].active = 0;
		}

		/* 1.3.5. Sanity check: No extra arguments at the end of line */
		if ( argc > focal_args+i*4+1 ) {
			fprintf(stderr,"-- Error in arguments: extra arguments at the end of the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
	}


#ifdef DEBUG
	/* 1.4. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d, max_iter: %d\n", rows, columns, max_iter);
	printf("Arguments, Teams: %d, Focal points: %d\n", num_teams, num_focal );
	for( i=0; i<num_teams; i++ ) {
		printf("\tTeam %d, position (%d,%d), type: %d\n", i, teams[i].x, teams[i].y, teams[i].type );
	}
	for( i=0; i<num_focal; i++ ) {
		printf("\tFocal_point %d, position (%d,%d), start time: %d, temperature: %d\n", i,
		focal[i].x,
		focal[i].y,
		focal[i].start,
		focal[i].heat );
	}
#endif // DEBUG

	/* 2. Select GPU and start global timer */
	hipSetDevice(0);
	hipDeviceSynchronize();
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */


	float *gpuSurface, *gpuSurfaceCopy, *gpuResiduals;
	int *gpuAux;
	FocalPoint *gpuFocal;
	Team *gpuTeam;
//	double time1,time2;
	int nearestUpperPow2 = pow(2,ceil(log2((double) rows*columns)));

	hipMalloc((void **)&gpuSurface,sizeof(float)*rows*columns);

	hipMalloc((void **)&gpuAux,sizeof(int)*rows*columns);


	hipMalloc((void **) &gpuSurfaceCopy,sizeof(float)*rows*columns);

	hipMalloc((void **) &gpuResiduals,sizeof(float)*nearestUpperPow2);


hipMalloc((void **) &gpuTeam,sizeof(Team)*num_teams);

	hipMemcpy(gpuTeam,teams,sizeof(Team)*num_teams,hipMemcpyHostToDevice);

	hipMalloc((void **) &gpuFocal,sizeof(FocalPoint)*num_focal);
	hipMemcpy(gpuFocal,focal,sizeof(FocalPoint)*num_focal,hipMemcpyHostToDevice);

	int tamBlockX= 128;
	int tamBlockY= 1;
	int tamGridX, tamGridY;
	int tamBlockTeams=224;
	int tamGridTeams;
	int tamBlockFocal=224;
	int tamGridFocal;

	tamGridTeams= num_teams/tamBlockTeams;
	if (num_teams%tamBlockTeams!=0) tamGridTeams++;

	tamGridFocal= num_focal/tamBlockFocal;
	if (num_focal%tamBlockFocal!=0) tamGridFocal++;

	tamGridX= columns/tamBlockX;
	if (columns%tamBlockX!=0) tamGridX++;
	tamGridY= rows/tamBlockY;
	if (rows%tamBlockY!=0) tamGridY++;

	dim3 blockSize(tamBlockX,tamBlockY);
	dim3 gridSize(tamGridX,tamGridY);
	#ifdef DEBUG
	printf("tamGx %d tamGy %d\n",tamGridX,tamGridY);
	#endif

	init<<<blockSize,gridSize>>>(gpuSurface,rows,columns);

	//CUDA_CHECK();
	init<<<blockSize,gridSize>>>(gpuSurfaceCopy,rows,columns);

	//CUDA_CHECK();
	/* 3. Initialize surface */
	/*for( i=0; i<rows; i++ )
		for( j=0; j<columns; j++ )
			accessMat( surface, i, j ) = 0.0;

	/* 4. Simulation */
	int *gpuNum_deactivated;
	//gpuNum_deactivated[0]=0;
	hipHostMalloc((void**) &gpuNum_deactivated,sizeof(int));
	int iter;
	int flag_stability = 0;
	//int first_activation = 0;
	//int *gpuFirstActivation;

	//hipHostMalloc((void**) &gpuFirstActivation,sizeof(int));
	//check_first_activation<<<tamGridFocal,tamBlockFocal>>>(gpuFocal,num_focal); hace falta reduccion


	//get_first_activation<<<tamGridFocal,tamBlockFocal>>>(gpuFocal,num_focal,gpuFirstActivation);
	#pragma unroll
	for( iter=0; iter<max_iter && ! flag_stability; iter++ ) {
		//printf("iter %d\n",iter);
		/* 4.1. Activate focal points */
		//printf("num %d\n",gpuNum_deactivated[0] );
		//hipMemcpy(gpuNum_deactivated,&num_deactivated,sizeof(int),hipMemcpyHostToDevice);
		//printf("num %d\n",num_deactivated);
		if(gpuNum_deactivated[0]<num_focal){
		activate_focal<<<tamGridFocal,tamBlockFocal>>>(gpuFocal,num_focal,gpuNum_deactivated,iter);
			hipDeviceSynchronize();
			//hipMemcpyAsync(&num_deactivated,gpuNum_deactivated,sizeof(int),hipMemcpyDeviceToHost,0);
		}
		//printf("num %d",num_deactivated);
		//if(!first_activation) continue;
		/* 4.2. Propagate heat (10 steps per each team movement) */
		float global_residual;
		int step;

		//hipMemcpy(surfaceCopy,gpuSurfaceCopy,sizeof(float)*rows*columns,hipMemcpyDeviceToHost);
		#pragma unroll
		for( step=0; step<10; step++ )	{
			/* 4.2.1. Update heat on active focal points */
				//if(gpuNum_deactivated[0]<num_focal)
				update_heat<<<tamGridFocal,tamBlockFocal>>>(gpuSurface,gpuFocal,columns,num_focal);

				//CUDA_CHECK();
				//accessMat( surface, x, y ) = focal[i].heat;



			/* 4.2.2. Copy values of the surface in ancillary structure (Skip borders) */
			//copy_surface<<<gridSize,blockSize>>>(gpuSurface,gpuSurfaceCopy,rows,columns);
		//	error=hipGetLastError();
		//	if(error!= hipSuccess)
		//		printf("%s\n",hipGetErrorString(error));
			float *aux=gpuSurface;
			gpuSurface=gpuSurfaceCopy;
			gpuSurfaceCopy=aux;
		//CUDA_CHECK();
			/*for( i=1; i<rows-1; i++ )
				for( j=1; j<columns-1; j++ )
					accessMat( surfaceCopy, i, j ) = accessMat( surface, i, j );

			/* 4.2.3. Update surface values (skip borders) */
			update_surface<<<gridSize,blockSize>>>(gpuSurface,gpuSurfaceCopy,rows,columns);

			//CUDA_CHECK();
			/*for( i=1; i<rows-1; i++ )
				for( j=1; j<columns-1; j++ )
					accessMat( surface, i, j ) = (
						accessMat( surfaceCopy, i-1, j ) +
						accessMat( surfaceCopy, i+1, j ) +
						accessMat( surfaceCopy, i, j-1 ) +
						accessMat( surfaceCopy, i, j+1 ) ) / 4;
			/* 4.2.4. Compute the maximum residual difference (absolute value) */

			if(step==0 && gpuNum_deactivated[0]==num_focal){
				//time1=cp_Wtime();
				//init<<<blockSize,gridSize>>>(gpuResiduals,rows,columns);
				compute_residual<<<gridSize,blockSize>>>(gpuSurface,gpuSurfaceCopy,rows,columns,gpuResiduals);

				//int numValues = nearestUpperPow2;
				int redSize = nearestUpperPow2;
				int blockSizeR = (1024);
				int sharedMemorySize = blockSizeR * sizeof(float);
				while ( redSize > 1 )
				{
					int baseNumBlocks = redSize/blockSizeR;

					int additionalBlock;
					if(redSize%blockSizeR==0)
						additionalBlock = 0;
					else
						additionalBlock = 1;

					int numBlocks = baseNumBlocks + additionalBlock;
					//printf("numB %d size %d\n",numBlocks,redSize);
					//if(numBlocks==1) exit(0);
					reduce_kernel<<< numBlocks, blockSizeR, sharedMemorySize >>>(gpuResiduals, gpuResiduals, redSize);
					redSize = numBlocks;
				}
				hipMemcpyAsync(&global_residual, gpuResiduals, sizeof(float), hipMemcpyDeviceToHost,0);

				//printf("glob %f\n",global_residual);

				//	printf("reesiduo %f\n",global_residual);
				//time2+=cp_Wtime()-time1;
			}

}

		/* If the global residual is lower than THRESHOLD, we have reached enough stability, stop simulation at the end of this iteration */

		/* 4.3. Move teams */
		if(gpuNum_deactivated[0]<num_focal){

			move_teams<<<tamGridTeams,tamBlockTeams>>>(gpuTeam,gpuFocal,num_teams,num_focal);

		}
		/* 4.4. Team actions */
		//hipMemcpy(surface,gpuSurface,sizeof(float)*rows*columns,hipMemcpyDeviceToHost);
		//initInt<<<gridSize,blockSize>>>()
		compute_heat_reduction<<<tamGridTeams,tamBlockTeams>>>(gpuTeam,gpuAux,num_teams,rows,columns);

		#ifdef UNROLL
		int *aux;
		aux = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );

		hipMemcpy(aux,gpuAux,sizeof(int)*rows*columns,hipMemcpyDeviceToHost);
		for( i=0;i<rows;i++){

		for( j=0;j<columns;j++)
			printf("%d ",aux[i*columns+j]);
			printf("\n" );
		}
		exit(0);
		#endif

		reduce_heat3<<<gridSize,blockSize>>>(gpuSurface,gpuAux,rows,columns);


		#ifdef DEBUG
				/* 4.5. DEBUG: Print the current state of the simulation at the end of each iteration */
				hipMemcpy(teams,gpuTeam,sizeof(Team)*num_teams,hipMemcpyDeviceToHost);
				hipMemcpy(surface,gpuSurface,sizeof(float)*rows*columns,hipMemcpyDeviceToHost);



				print_status( iter, rows, columns, surface, num_teams, teams, num_focal, focal, global_residual );
		#endif // DEBUG

	if( gpuNum_deactivated[0] == num_focal && global_residual < THRESHOLD ) flag_stability = 1;
	}
	hipMemcpy(surface,gpuSurface,sizeof(float)*rows*columns,hipMemcpyDeviceToHost);

		//hipMemcpy(focal,gpuFocal,sizeof(FocalPoint)*num_focal,hipMemcpyDeviceToHost);
//hipFree(gpuSurface);
//hipFree(gpuSurfaceCopy);
//hipFree(gpuTeam);
//hipFree(gpuFocal);
//printf("time1 %f\n",time2);
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 5. Stop global time */
	hipDeviceSynchronize();
	ttotal = cp_Wtime() - ttotal;

	/* 6. Output for leaderboard */
	printf("\n");
	/* 6.1. Total computation time */
	printf("Time: %lf\n", ttotal );
	/* 6.2. Results: Number of iterations, position of teams, residual heat on the focal points */
	printf("Result: %d", iter);
	/*
	for (i=0; i<num_teams; i++)
		printf(" %d %d", teams[i].x, teams[i].y );
	*/
	for (i=0; i<num_focal; i++)
		printf(" %.6f", accessMat( surface, focal[i].x, focal[i].y ) );
	printf("\n");

	/* 7. Free resources */
	free( teams );
	free( focal );
	free( surface );
	free( surfaceCopy );

	/* 8. End */
	return 0;
}
