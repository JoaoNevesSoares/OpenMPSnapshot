#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "Solve_Systems.cuh"

// Per assicurarmi di non eccedere il limite dei 1024 blocchi della "Tesla T4"
#define TILE 16



/************************************************************************/
/*******| Funzione per effettuare la Decomposizione LU sulla GPU |*******/
/************************************************************************/

void LUDecompose(double* gpu_a, int n, int numblock) {

    // Itera attraverso le colonne della matrice
    for (int i = 0; i < n; ++i) {

        scala_Indice<<<1, 1>>>(gpu_a, n, i);
        eliminazione_gaussiana<<<numblock, TILE, n * sizeof(double)>>>(gpu_a, n, i, TILE);

    }

}


// Kernel CUDA per scalare la riga corrente
__global__ void scala_Indice(double *matrix, int n, int index) {

    int start = (index * n + index);
    int end = (index * n + n);

    // Normalizzazione Doolittle (LU) --> Divide ogni elemento della riga per il pivot
    for (int i = start + 1; i < end; ++i) {
        matrix[i] = (matrix[i] / matrix[start]);
    }

}


// Kernel CUDA per l'eliminazione gaussiana
__global__ void eliminazione_gaussiana(double *A, int n, int index, int bsize) {

    extern __shared__ double pivot[];

    int idThread = threadIdx.x;
    int idBlock = blockIdx.x;
    int blockSize = bsize;

    // Copia il pivot nella memoria condivisa
    if (idThread == 0) {

        for (int i = index; i < n; i++) pivot[i] = A[(index * n) + i];

    }


    // Aspetto che tutti i thread del blocco siano terminati
    __syncthreads();


    int pivotRow = (index * n);
    int currentRow = (((blockSize * idBlock) + idThread) * n);
    int start = currentRow + index;
    int end = currentRow + n;

    // Esegue l'eliminazione gaussiana sui blocchi paralleli
    if (currentRow > pivotRow) {

        for (int i = start + 1; i < end; ++i) {

            A[i] = A[i] - (A[start] * pivot[i - currentRow]);

        }

    }

}




/*************************************************************************/
/*******|  Funzione per risolvere il sistema lineare (sulla CPU)  |*******/
/*************************************************************************/

int LUSolve(int n, double** L, double** U, double* b) {

    // Imposta il numero di core della CPU utilizzati per la parallelizzazione
    int NUM_CORES = omp_get_num_procs();
    omp_set_num_threads(NUM_CORES);


    // Forward substitution (Ly = b)
    #pragma omp parallel for
    for (int i = 0; i < n; i++) {
        double sum = b[i];
        for (int j = 0; j < i; j++) {
            sum -= L[i][j] * b[j];
        }
        b[i] = sum / L[i][i];
    }


    // Backward substitution (Ux = y)
    #pragma omp parallel for
    for (int i = n - 1; i >= 0; i--) {
        double sum = b[i];
        for (int j = i + 1; j < n; j++) {
            sum -= U[i][j] * b[j];
        }
        b[i] = sum / U[i][i];
    }


    return NUM_CORES;

}




/************************************************************************/
/*******|  Funzione di utilità per gestire le matrici dinamiche  |*******/
/************************************************************************/

void generaMatrice(double* a, int n) {

    // Inizializza la matrice con numeri casuali compresi tra -100 e 100
    for (int i = 0; i <= (n * n); ++i) {
        a[i] = ((rand() % 201) - 100);
    }

    int diagCount = 0;
    double sum = 0;

    // Imposta i valori sulla diagonale in modo che la matrice sia diagonale dominante
    for (int i = 0; i < n; ++i) {
        for (int j = i * n; j < i * n + n; ++j) {
            sum += abs(a[j]);
        }
        sum -= abs(a[i * n + diagCount]);
        a[i * n + diagCount] = sum + ((rand() % 5) + 1);
        ++diagCount;
        sum = 0;
    }

}


void initialize_matrices(double** a, double** l, double** u, int size) {

    for (int i = 0; i < size; ++i) {
        a[i] = (double*)malloc(size * sizeof(double));
        l[i] = (double*)malloc(size * sizeof(double));
        u[i] = (double*)malloc(size * sizeof(double));
    }

}


void deallocate_matrices(double** a, double** l, double** u, int size) {

    for (int i = 0; i < size; ++i) {
        free(a[i]);
        free(l[i]);
        free(u[i]);
    }

    free(a);
    free(l);
    free(u);

}