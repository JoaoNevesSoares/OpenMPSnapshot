#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include <omp.h>

#define BLOCK_SIZE 256
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.01 
#define SOFTENING 1e-9f  /* Will guard against denormals */

typedef struct { float4 *pos, *vel; } Body;

void checkCudaErrors(hipError_t error){
	if(error != hipSuccess) {
		printf("\033[0;31mCUDA Error: %s in %s, line %d\033[0;37m\n", hipGetErrorString(error), __FILE__, __LINE__);
	}
}

void randomizeBodies(float *data, int n) {
 srand(100);
 for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

void bodyForce(float4 *p, float4 *v, float dt, int n) {
	#pragma omp parallel for
  for (int i = 0; i < n; i++) { 
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = 1.0f / sqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    v[i].x += dt*Fx; v[i].y += dt*Fy; v[i].z += dt*Fz;
  }

}

__global__ void d_bodyForce(float4 *p, float4 *v, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
    
    for (int index = 0; index < gridDim.x; index++) {
      __shared__ float4 shared_pos[BLOCK_SIZE];
      shared_pos[threadIdx.x] = p[index * blockDim.x + threadIdx.x];
      __syncthreads();
	  
      for (int j = 0; j < BLOCK_SIZE; j++) {
        float dx = shared_pos[j].x - p[i].x;
        float dy = shared_pos[j].y - p[i].y;
        float dz = shared_pos[j].z - p[i].z;
        float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
      }
      __syncthreads();
    }

    v[i].x += dt*Fx; v[i].y += dt*Fy; v[i].z += dt*Fz;
  }
}

int main(const int argc, const char** argv) {

  int nBodies = 30000;
  if (argc > 1) nBodies = atoi(argv[1]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int bytes = 2*nBodies*sizeof(float4);
  //we save the gpu output to these 
  float *buf = (float*)malloc(bytes); 
  Body p = { (float4*)buf, ((float4*)buf) + nBodies };
  
  //they are used for comparison
  float *h_buf = (float*)malloc(bytes);
  Body h_p = { (float4*)h_buf, ((float4*)h_buf) + nBodies };

  randomizeBodies(buf, 8*nBodies); // Init pos / vel data
  randomizeBodies(h_buf, 8*nBodies); // Init pos / vel data
  
  //GPU initialisation
  float *d_buf;
  hipMalloc(&d_buf, bytes);
  checkCudaErrors(hipGetLastError());
  Body d_p = { (float4*)d_buf, ((float4*)d_buf) + nBodies };
  
  dim3 grid_dim(ceil((double)nBodies / BLOCK_SIZE));

  double totalTime = 0.0;

  for (int iter = 1; iter <= nIters; iter++) {
    if (iter == 1) {
		bodyForce(h_p.pos, h_p.vel, dt, nBodies); // compute interbody forces
		for (int i = 0 ; i < nBodies; i++) { // integrate position
		   h_p.pos[i].x += h_p.vel[i].x*dt;
      	   h_p.pos[i].y += h_p.vel[i].y*dt;
      	   h_p.pos[i].z += h_p.vel[i].z*dt;
		}
	printf("CPU CALCULATIONS ENDED\n");
	}
    StartTimer();

	hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());
    d_bodyForce<<<grid_dim, BLOCK_SIZE>>>(d_p.pos, d_p.vel, dt, nBodies); // compute interbody forces
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p.pos[i].x += p.vel[i].x*dt;
      p.pos[i].y += p.vel[i].y*dt;
      p.pos[i].z += p.vel[i].z*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    if (iter == 1) {
		for (int i = 0 ; i < nBodies; i++) {
			if (ABS(h_p.pos[i].x - p.pos[i].x) >= accuracy  || ABS (h_p.pos[i].y - p.pos[i].y) >= accuracy || ABS (h_p.pos[i].z - p.pos[i].z) >= accuracy) {
				printf("ERORR!\n");
				free(buf);
				free(h_buf);
	  			hipFree(d_buf);
	  			hipDeviceReset();
				return 0;
			}
		}
	printf("COMPARISONS ENDED SUCCESFULLY\n");
	}
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
    }
    printf("Iteration %d: %.3f seconds\n", iter, tElapsed);
  }
  double avgTime = totalTime / (double)(nIters-1); 

  printf("%d Bodies: average %0.3f Billion Interactions / second\nAVERAGE TIME %.3f\n", nBodies, 1e-9 * nBodies * nBodies / avgTime, avgTime);
  free(buf);
  free(h_buf);
  hipFree(d_buf);
  hipDeviceReset();
}
